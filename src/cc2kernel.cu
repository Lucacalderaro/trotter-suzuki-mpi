#include "hip/hip_runtime.h"
/**
 * Distributed Trotter-Suzuki solver
 * Copyright (C) 2015 Luca Calderaro, 2012-2015 Peter Wittek,
 * 2010-2012 Carlos Bederián
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#undef _GLIBCXX_ATOMIC_BUILTINS
#include <cassert>
#include <vector>
#include <map>
#include <stdio.h>

#if HAVE_CONFIG_H
#include <config.h>
#endif
#include "cc2kernel.h"
#include "common.h"
#ifdef HAVE_MPI
#include <mpi.h>
#endif

/** Check and initialize a device attached to a node
 *  @param commRank - the MPI rank of this process
 *  @param commSize - the size of MPI comm world
 *  This snippet is from GPMR:
 *  http://code.google.com/p/gpmr/
 */
void setDevice(int commRank
#ifdef HAVE_MPI
               , MPI_Comm cartcomm
#endif
              ) {
    int commSize = 1;
    int devCount;
    int deviceNum = 0; //-1;
    CUDA_SAFE_CALL(hipGetDeviceCount(&devCount));
    
#ifdef HAVE_MPI
    MPI_Comm_size(cartcomm, &commSize);
#ifdef _WIN32
	FILE * fp = popen("hostname.exe", "r");
#else
	FILE * fp = popen("/bin/hostname", "r");
#endif
    char buf[1024];
    if (fgets(buf, 1023, fp) == NULL) strcpy(buf, "localhost");
    pclose(fp);
    std::string host = buf;
    host = host.substr(0, host.size() - 1);
    strcpy(buf, host.c_str());

    if (commRank == 0) {
        std::map<std::string, std::vector<int> > hosts;
        std::map<std::string, int> devCounts;
        MPI_Status stat;
        MPI_Request req;

        hosts[buf].push_back(0);
        devCounts[buf] = devCount;
        for (int i = 1; i < commSize; ++i) {
            MPI_Recv(buf, 1024, MPI_CHAR, i, 0, cartcomm, &stat);
            MPI_Recv(&devCount, 1, MPI_INT, i, 0, cartcomm, &stat);

            // check to make sure each process on each node reports the same number of devices.
            hosts[buf].push_back(i);
            if (devCounts.find(buf) != devCounts.end()) {
                if (devCounts[buf] != devCount) {
                    printf("Error, device count mismatch %d != %d on %s\n", devCounts[buf], devCount, buf);
                    fflush(stdout);
                }
            }
            else devCounts[buf] = devCount;
        }
        // check to make sure that we don't have more jobs on a node than we have GPUs.
        for (std::map<std::string, std::vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            if (it->second.size() > static_cast<unsigned int>(devCounts[it->first])) {
                printf("Error, more jobs running on '%s' than devices - %d jobs > %d devices.\n",
                       it->first.c_str(), static_cast<int>(it->second.size()), devCounts[it->first]);
                fflush(stdout);
                MPI_Abort(cartcomm, 1);
            }
        }

        // send out the device number for each process to use.
        MPI_Irecv(&deviceNum, 1, MPI_INT, 0, 0, cartcomm, &req);
        for (std::map<std::string, std::vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            for (unsigned int i = 0; i < it->second.size(); ++i) {
                int devID = i;
                MPI_Send(&devID, 1, MPI_INT, it->second[i], 0, cartcomm);
            }
        }
        MPI_Wait(&req, &stat);
    }
    else {
        // send out the hostname and device count for your local node, then get back the device number you should use.
        MPI_Status stat;
        MPI_Send(buf, strlen(buf) + 1, MPI_CHAR, 0, 0, cartcomm);
        MPI_Send(&devCount, 1, MPI_INT, 0, 0, cartcomm);
        MPI_Recv(&deviceNum, 1, MPI_INT, 0, 0, cartcomm, &stat);
    }
    MPI_Barrier(cartcomm);
#endif
    CUDA_SAFE_CALL(hipSetDevice(deviceNum));
}

//REAL TIME functions

template<int BLOCK_WIDTH, int BLOCK_HEIGHT, int BACKWARDS>
inline __device__ void trotter_vert_pair_flexible_nosync(double a, double b, int tile_height, double &cell_r, double &cell_i, int kx, int ky, int py, double rl[BLOCK_HEIGHT][BLOCK_WIDTH], double im[BLOCK_HEIGHT][BLOCK_WIDTH]) {
    double peer_r;
    double peer_i;

    const int ky_peer = ky + 1 - 2 * BACKWARDS;
    if (py >= BACKWARDS && py < tile_height - 1 + BACKWARDS && ky >= BACKWARDS && ky < BLOCK_HEIGHT - 1 + BACKWARDS) {
        peer_r = rl[ky_peer][kx];
        peer_i = im[ky_peer][kx];
#ifndef DISABLE_FMA
        rl[ky_peer][kx] = a * peer_r - b * cell_i;
        im[ky_peer][kx] = a * peer_i + b * cell_r;
        cell_r = a * cell_r - b * peer_i;
        cell_i = a * cell_i + b * peer_r;
#else
        // NOTE: disabling FMA has worse precision and performance
        //       use only for exact implementation verification against CPU results
        rl[ky_peer][kx] = __dadd_rn(a * peer_r, - b * cell_i);
        im[ky_peer][kx] = __dadd_rn(a * peer_i, b * cell_r);
        cell_r = __dadd_rn(a * cell_r, - b * peer_i);
        cell_i = __dadd_rn(a * cell_i, b * peer_r);
#endif
    }
}


template<int BLOCK_WIDTH, int BLOCK_HEIGHT, int BACKWARDS>
static  inline __device__ void trotter_horz_pair_flexible_nosync(double a, double b,  int tile_width, double &cell_r, double &cell_i, int kx, int ky, int px, double rl[BLOCK_HEIGHT][BLOCK_WIDTH], double im[BLOCK_HEIGHT][BLOCK_WIDTH]) {
    double peer_r;
    double peer_i;

    const int kx_peer = kx + 1 - 2 * BACKWARDS;
    if (px >= BACKWARDS && px < tile_width - 1 + BACKWARDS && kx >= BACKWARDS && kx < BLOCK_WIDTH - 1 + BACKWARDS) {
        peer_r = rl[ky][kx_peer];
        peer_i = im[ky][kx_peer];
#ifndef DISABLE_FMA
        rl[ky][kx_peer] = a * peer_r - b * cell_i;
        im[ky][kx_peer] = a * peer_i + b * cell_r;
        cell_r = a * cell_r - b * peer_i;
        cell_i = a * cell_i + b * peer_r;
#else
        // NOTE: disabling FMA has worse precision and performance
        //       use only for exact implementation verification against CPU results
        rl[ky][kx_peer] = __dadd_rn(a * peer_r, - b * cell_i);
        im[ky][kx_peer] = __dadd_rn(a * peer_i, b * cell_r);
        cell_r = __dadd_rn(a * cell_r, - b * peer_i);
        cell_i = __dadd_rn(a * cell_i, b * peer_r);
#endif
    }
}

template<int BLOCK_WIDTH, int BLOCK_HEIGHT>
static  inline __device__ void trotter_external_pot_nosync(int tile_width, int tile_height, double &cell_r, double &cell_i,
        int kx, int ky, int px, int py,
        double rl[BLOCK_HEIGHT][BLOCK_WIDTH], double im[BLOCK_HEIGHT][BLOCK_WIDTH],
        double pot_r[BLOCK_HEIGHT][BLOCK_WIDTH], double pot_i[BLOCK_HEIGHT][BLOCK_WIDTH]) {
    double var;
    double peer_r;
    double peer_i;
    double pot_cell_r, pot_cell_i, pot_peer_r, pot_peer_i;

    const int ky_peer = ky + 1 - 2 * (kx % 2);
    if(ky >= 0 && ky < BLOCK_HEIGHT && ky_peer >= 0 && ky_peer < BLOCK_HEIGHT && kx >= 0 && kx < BLOCK_WIDTH) {
        pot_cell_r = pot_r[ky][kx];
        pot_cell_i = pot_i[ky][kx];
        pot_peer_r = pot_r[ky_peer][kx];
        pot_peer_i = pot_i[ky_peer][kx];

        peer_r = rl[ky_peer][kx];
        peer_i = im[ky_peer][kx];

#ifndef DISABLE_FMA
        var = cell_r;
        cell_r = pot_cell_r * var - pot_cell_i * cell_i;
        cell_i = pot_cell_r * cell_i + pot_cell_i * var;

        rl[ky_peer][kx] = pot_peer_r * peer_r - pot_peer_i * peer_i;
        im[ky_peer][kx] = pot_peer_r * peer_i + pot_peer_i * peer_r;
#else
        // NOTE: disabling FMA has worse precision and performance
        //       use only for exact implementation verification against CPU results
        var = cell_r;
        cell_r = __dadd_rn(pot_cell_r * var, - pot_cell_i * cell_i);
        cell_i = __dadd_rn(pot_cell_r * cell_i, pot_cell_i * var);

        rl[ky_peer][kx] = __dadd_rn(pot_peer_r * peer_r, - pot_peer_i * peer_i);
        im[ky_peer][kx] = __dadd_rn(pot_peer_r * peer_i, pot_peer_i * peer_r);
#endif
    }
}

__launch_bounds__(BLOCK_X * STRIDE_Y)
__global__ void cc2kernel(size_t tile_width, size_t tile_height, size_t offset_x, size_t offset_y, size_t halo_x, size_t halo_y,
                          double a, double b, const double * __restrict__ external_pot_real, const double * __restrict__ external_pot_imag,
                          const double * __restrict__ p_real, const double * __restrict__ p_imag,
                          double * __restrict__ p2_real, double * __restrict__ p2_imag,
                          int inner, int horizontal, int vertical) {

    __shared__ double rl[BLOCK_Y][BLOCK_X];
    __shared__ double im[BLOCK_Y][BLOCK_X];
    __shared__ double pot_r[BLOCK_Y][BLOCK_X];
    __shared__ double pot_i[BLOCK_Y][BLOCK_X];

    int blockIdxx = inner * (blockIdx.x + 1) + horizontal * (blockIdx.x) + vertical * (blockIdx.x * ((tile_width + (BLOCK_X - 2 * halo_x) - 1) / (BLOCK_X - 2 * halo_x) - 1));
    int blockIdxy = inner * (blockIdx.y + 1) + horizontal * (blockIdx.y * ((tile_height + (BLOCK_Y - 2 * halo_y) - 1) / (BLOCK_Y - 2 * halo_y) - 1)) + vertical * (blockIdx.y + 1);

    // The offsets are used by the hybrid kernel
    int px = offset_x + blockIdxx * (BLOCK_X - 2 * halo_x) + threadIdx.x - halo_x;
    int py = offset_y + blockIdxy * (BLOCK_Y - 2 * halo_y) + threadIdx.y - halo_y;

    // Read block from global into shared memory (state and potential)
    if (px >= 0 && px < tile_width) {
#pragma unroll
        for (int i = 0, pidx = py * tile_width + px; i < BLOCK_Y / STRIDE_Y; ++i, pidx += STRIDE_Y * tile_width) {
            if (py + i * STRIDE_Y >= 0 && py + i * STRIDE_Y < tile_height) {
                rl[threadIdx.y + i * STRIDE_Y][threadIdx.x] = p_real[pidx];
                im[threadIdx.y + i * STRIDE_Y][threadIdx.x] = p_imag[pidx];
                pot_r[threadIdx.y + i * STRIDE_Y][threadIdx.x] = external_pot_real[pidx];
                pot_i[threadIdx.y + i * STRIDE_Y][threadIdx.x] = external_pot_imag[pidx];
            }
        }
    }

    __syncthreads();

    // Place threads along the black cells of a checkerboard pattern
    int sx = threadIdx.x;
    int sy;
    if ((halo_x) % 2 == (halo_y) % 2) {
        sy = 2 * threadIdx.y + threadIdx.x % 2;
    }
    else {
        sy = 2 * threadIdx.y + 1 - threadIdx.x % 2;
    }

    // global y coordinate of the thread on the checkerboard (px remains the same)
    // used for range checks
    int checkerboard_py = offset_y + blockIdxy * (BLOCK_Y - 2 * halo_y) + sy - halo_y;

    // Keep the fixed black cells on registers, reds are updated in shared memory
    double cell_r[BLOCK_Y / (STRIDE_Y * 2)];
    double cell_i[BLOCK_Y / (STRIDE_Y * 2)];

#pragma unroll
    // Read black cells to registers
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        cell_r[part] = rl[sy + part * 2 * STRIDE_Y][sx];
        cell_i[part] = im[sy + part * 2 * STRIDE_Y][sx];
    }

    // 12344321
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
//potential
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_external_pot_nosync<BLOCK_X, BLOCK_Y>(tile_width, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, checkerboard_py + part * 2 * STRIDE_Y, rl, im, pot_r, pot_i);
    }
    __syncthreads();

#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();


    // Write black cells in registers to shared memory
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        rl[sy + part * 2 * STRIDE_Y][sx] = cell_r[part];
        im[sy + part * 2 * STRIDE_Y][sx] = cell_i[part];
    }
    __syncthreads();

    // discard the halo and copy results from shared to global memory
    sx = threadIdx.x + halo_x;
    sy = threadIdx.y + halo_y;
    px += halo_x;
    py += halo_y;
    if (sx < BLOCK_X - halo_x && px < tile_width) {
#pragma unroll
        for (int i = 0, pidx = py * tile_width + px; i < BLOCK_Y / STRIDE_Y; ++i, pidx += STRIDE_Y * tile_width) {
            if (sy + i * STRIDE_Y < BLOCK_Y - halo_y && py + i * STRIDE_Y < tile_height) {
                p2_real[pidx] = rl[sy + i * STRIDE_Y][sx];
                p2_imag[pidx] = im[sy + i * STRIDE_Y][sx];
            }
        }
    }
}

//  IMAGINARY TIME functions

template<int BLOCK_WIDTH, int BLOCK_HEIGHT, int BACKWARDS>
inline __device__ void imag_trotter_vert_pair_flexible_nosync(double a, double b, int tile_height, double &cell_r, double &cell_i, int kx, int ky, int py, double rl[BLOCK_HEIGHT][BLOCK_WIDTH], double im[BLOCK_HEIGHT][BLOCK_WIDTH]) {
    double peer_r;
    double peer_i;

    const int ky_peer = ky + 1 - 2 * BACKWARDS;
    if (py >= BACKWARDS && py < tile_height - 1 + BACKWARDS && ky >= BACKWARDS && ky < BLOCK_HEIGHT - 1 + BACKWARDS) {
        peer_r = rl[ky_peer][kx];
        peer_i = im[ky_peer][kx];
#ifndef DISABLE_FMA
        rl[ky_peer][kx] = a * peer_r + b * cell_r;
        im[ky_peer][kx] = a * peer_i + b * cell_i;
        cell_r = a * cell_r + b * peer_r;
        cell_i = a * cell_i + b * peer_i;
#else
        // NOTE: disabling FMA has worse precision and performance
        //       use only for exact implementation verification against CPU results
        rl[ky_peer][kx] = __dadd_rn(a * peer_r, b * cell_r);
        im[ky_peer][kx] = __dadd_rn(a * peer_i, b * cell_i);
        cell_r = __dadd_rn(a * cell_r, b * peer_r);
        cell_i = __dadd_rn(a * cell_i, b * peer_i);
#endif
    }
}


template<int BLOCK_WIDTH, int BLOCK_HEIGHT, int BACKWARDS>
static  inline __device__ void imag_trotter_horz_pair_flexible_nosync(double a, double b,  int tile_width, double &cell_r, double &cell_i, int kx, int ky, int px, double rl[BLOCK_HEIGHT][BLOCK_WIDTH], double im[BLOCK_HEIGHT][BLOCK_WIDTH]) {
    double peer_r;
    double peer_i;

    const int kx_peer = kx + 1 - 2 * BACKWARDS;
    if (px >= BACKWARDS && px < tile_width - 1 + BACKWARDS && kx >= BACKWARDS && kx < BLOCK_WIDTH - 1 + BACKWARDS) {
        peer_r = rl[ky][kx_peer];
        peer_i = im[ky][kx_peer];
#ifndef DISABLE_FMA
        rl[ky][kx_peer] = a * peer_r + b * cell_r;
        im[ky][kx_peer] = a * peer_i + b * cell_i;
        cell_r = a * cell_r + b * peer_r;
        cell_i = a * cell_i + b * peer_i;
#else
        // NOTE: disabling FMA has worse precision and performance
        //       use only for exact implementation verification against CPU results
        rl[ky][kx_peer] = __dadd_rn(a * peer_r, b * cell_r);
        im[ky][kx_peer] = __dadd_rn(a * peer_i, b * cell_i);
        cell_r = __dadd_rn(a * cell_r, b * peer_r);
        cell_i = __dadd_rn(a * cell_i, b * peer_i);
#endif
    }
}

template<int BLOCK_WIDTH, int BLOCK_HEIGHT>
static  inline __device__ void imag_trotter_external_pot_nosync(int tile_width, int tile_height, double &cell_r, double &cell_i,
        int kx, int ky, int px, int py,
        double rl[BLOCK_HEIGHT][BLOCK_WIDTH], double im[BLOCK_HEIGHT][BLOCK_WIDTH],
        double pot_r[BLOCK_HEIGHT][BLOCK_WIDTH]) {
    double peer_r;
    double peer_i;
    double pot_cell_r, pot_peer_r;

    const int ky_peer = ky + 1 - 2 * (kx % 2);
    if(ky >= 0 && ky < BLOCK_HEIGHT && ky_peer >= 0 && ky_peer < BLOCK_HEIGHT && kx >= 0 && kx < BLOCK_WIDTH) {
        pot_cell_r = pot_r[ky][kx];
        pot_peer_r = pot_r[ky_peer][kx];
        peer_r = rl[ky_peer][kx];
        peer_i = im[ky_peer][kx];

        cell_r = pot_cell_r * cell_r;
        cell_i = pot_cell_r * cell_i;
        rl[ky_peer][kx] = pot_peer_r * peer_r;
        im[ky_peer][kx] = pot_peer_r * peer_i;
    }
}

__launch_bounds__(BLOCK_X * STRIDE_Y)
__global__ void imag_cc2kernel(size_t tile_width, size_t tile_height, size_t offset_x, size_t offset_y, size_t halo_x, size_t halo_y,
                               double a, double b, const double * __restrict__ external_pot_real, const double * __restrict__ external_pot_imag,
                               const double * __restrict__ p_real, const double * __restrict__ p_imag,
                               double * __restrict__ p2_real, double * __restrict__ p2_imag,
                               int inner, int horizontal, int vertical) {

    __shared__ double rl[BLOCK_Y][BLOCK_X];
    __shared__ double im[BLOCK_Y][BLOCK_X];
    __shared__ double pot_r[BLOCK_Y][BLOCK_X];

    int blockIdxx = inner * (blockIdx.x + 1) + horizontal * (blockIdx.x) + vertical * (blockIdx.x * ((tile_width + (BLOCK_X - 2 * halo_x) - 1) / (BLOCK_X - 2 * halo_x) - 1));
    int blockIdxy = inner * (blockIdx.y + 1) + horizontal * (blockIdx.y * ((tile_height + (BLOCK_Y - 2 * halo_y) - 1) / (BLOCK_Y - 2 * halo_y) - 1)) + vertical * (blockIdx.y + 1);

    // The offsets are used by the hybrid kernel
    int px = offset_x + blockIdxx * (BLOCK_X - 2 * halo_x) + threadIdx.x - halo_x;
    int py = offset_y + blockIdxy * (BLOCK_Y - 2 * halo_y) + threadIdx.y - halo_y;

    // Read block from global into shared memory (state and potential)
    if (px >= 0 && px < tile_width) {
#pragma unroll
        for (int i = 0, pidx = py * tile_width + px; i < BLOCK_Y / STRIDE_Y; ++i, pidx += STRIDE_Y * tile_width) {
            if (py + i * STRIDE_Y >= 0 && py + i * STRIDE_Y < tile_height) {
                rl[threadIdx.y + i * STRIDE_Y][threadIdx.x] = p_real[pidx];
                im[threadIdx.y + i * STRIDE_Y][threadIdx.x] = p_imag[pidx];
                pot_r[threadIdx.y + i * STRIDE_Y][threadIdx.x] = external_pot_real[pidx];
            }
        }
    }

    __syncthreads();

    // Place threads along the black cells of a checkerboard pattern
    int sx = threadIdx.x;
    int sy;
    if ((halo_x) % 2 == (halo_y) % 2) {
        sy = 2 * threadIdx.y + threadIdx.x % 2;
    }
    else {
        sy = 2 * threadIdx.y + 1 - threadIdx.x % 2;
    }

    // global y coordinate of the thread on the checkerboard (px remains the same)
    // used for range checks
    int checkerboard_py = offset_y + blockIdxy * (BLOCK_Y - 2 * halo_y) + sy - halo_y;

    // Keep the fixed black cells on registers, reds are updated in shared memory
    double cell_r[BLOCK_Y / (STRIDE_Y * 2)];
    double cell_i[BLOCK_Y / (STRIDE_Y * 2)];

#pragma unroll
    // Read black cells to registers
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        cell_r[part] = rl[sy + part * 2 * STRIDE_Y][sx];
        cell_i[part] = im[sy + part * 2 * STRIDE_Y][sx];
    }

    // 12344321
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
//potential
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_external_pot_nosync<BLOCK_X, BLOCK_Y>(tile_width, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, checkerboard_py + part * 2 * STRIDE_Y, rl, im, pot_r);
    }
    __syncthreads();

#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 1>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_horz_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_width, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, px, rl, im);
    }
    __syncthreads();
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        imag_trotter_vert_pair_flexible_nosync<BLOCK_X, BLOCK_Y, 0>(a, b, tile_height, cell_r[part], cell_i[part], sx, sy + part * 2 * STRIDE_Y, checkerboard_py + part * 2 * STRIDE_Y, rl, im);
    }
    __syncthreads();


    // Write black cells in registers to shared memory
#pragma unroll
    for (int part = 0; part < BLOCK_Y / (STRIDE_Y * 2); ++part) {
        rl[sy + part * 2 * STRIDE_Y][sx] = cell_r[part];
        im[sy + part * 2 * STRIDE_Y][sx] = cell_i[part];
    }
    __syncthreads();

    // discard the halo and copy results from shared to global memory
    sx = threadIdx.x + halo_x;
    sy = threadIdx.y + halo_y;
    px += halo_x;
    py += halo_y;
    if (sx < BLOCK_X - halo_x && px < tile_width) {
#pragma unroll
        for (int i = 0, pidx = py * tile_width + px; i < BLOCK_Y / STRIDE_Y; ++i, pidx += STRIDE_Y * tile_width) {
            if (sy + i * STRIDE_Y < BLOCK_Y - halo_y && py + i * STRIDE_Y < tile_height) {
                p2_real[pidx] = rl[sy + i * STRIDE_Y][sx];
                p2_imag[pidx] = im[sy + i * STRIDE_Y][sx];
            }
        }
    }
}

// Wrapper function for the hybrid kernel
void cc2kernel_wrapper(size_t tile_width, size_t tile_height, size_t offset_x, size_t offset_y, size_t halo_x, size_t halo_y, dim3 numBlocks, dim3 threadsPerBlock, hipStream_t stream, double a, double b, const double * __restrict__ dev_external_pot_real, const double * __restrict__ dev_external_pot_imag, const double * __restrict__ pdev_real, const double * __restrict__ pdev_imag, double * __restrict__ pdev2_real, double * __restrict__ pdev2_imag, int inner, int horizontal, int vertical, bool imag_time) {
    if(imag_time)
        imag_cc2kernel <<< numBlocks, threadsPerBlock, 0, stream>>>(tile_width, tile_height, offset_x, offset_y, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real, pdev_imag, pdev2_real, pdev2_imag, inner, horizontal, vertical);
    else
        cc2kernel <<< numBlocks, threadsPerBlock, 0, stream>>>(tile_width, tile_height, offset_x, offset_y, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real, pdev_imag, pdev2_real, pdev2_imag, inner, horizontal, vertical);
    CUT_CHECK_ERROR("Kernel error in cc2kernel_wrapper");
}

CC2Kernel::CC2Kernel(double *_p_real, double *_p_imag, double *_external_pot_real, double *_external_pot_imag, double _a, double _b, int matrix_width, int matrix_height, int _halo_x, int _halo_y, int *_periods, bool _imag_time
#ifdef HAVE_MPI
                     , MPI_Comm _cartcomm
#endif
                     ):
    p_real(_p_real),
    p_imag(_p_imag),
    external_pot_real(_external_pot_real),
    external_pot_imag(_external_pot_imag),
    threadsPerBlock(BLOCK_X, STRIDE_Y),
    sense(0),
    a(_a),
    b(_b),
    halo_x(_halo_x),
    halo_y(_halo_y),
    imag_time(_imag_time) {

    periods = _periods;
    int rank, coords[2], dims[2] = {0, 0};
#ifdef HAVE_MPI
    cartcomm = _cartcomm;
    MPI_Cart_shift(cartcomm, 0, 1, &neighbors[UP], &neighbors[DOWN]);
    MPI_Cart_shift(cartcomm, 1, 1, &neighbors[LEFT], &neighbors[RIGHT]);
    MPI_Comm_rank(cartcomm, &rank);
    MPI_Cart_get(cartcomm, 2, dims, periods, coords);
#else
    neighbors[UP] = neighbors[DOWN] = neighbors[LEFT] = neighbors[RIGHT] = 0;
    dims[0] = dims[1] = 1;
    rank = 0;
    coords[0] = coords[1] = 0;
#endif
    int inner_start_x = 0, end_x = 0, end_y = 0;
    calculate_borders(coords[1], dims[1], &start_x, &end_x, &inner_start_x, &inner_end_x, matrix_width - 2 * periods[1]*halo_x, halo_x, periods[1]);
    calculate_borders(coords[0], dims[0], &start_y, &end_y, &inner_start_y, &inner_end_y, matrix_height - 2 * periods[0]*halo_y, halo_y, periods[0]);
    tile_width = end_x - start_x;
    tile_height = end_y - start_y;
    
    setDevice(rank
#ifdef HAVE_MPI
              , cartcomm
#endif
              );
              
    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&dev_external_pot_real), tile_width * tile_height * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&dev_external_pot_imag), tile_width * tile_height * sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpy(dev_external_pot_real, external_pot_real, tile_width * tile_height * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_external_pot_imag, external_pot_imag, tile_width * tile_height * sizeof(double), hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&pdev_real[0]), tile_width * tile_height * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&pdev_real[1]), tile_width * tile_height * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&pdev_imag[0]), tile_width * tile_height * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&pdev_imag[1]), tile_width * tile_height * sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpy(pdev_real[0], p_real, tile_width * tile_height * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(pdev_imag[0], p_imag, tile_width * tile_height * sizeof(double), hipMemcpyHostToDevice));
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Halo exchange uses wave pattern to communicate
    int height = inner_end_y - inner_start_y;	// The vertical halo in rows
    int width = halo_x;	// The number of columns of the matrix
    // Allocating pinned memory for the buffers
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &left_real_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &left_real_send, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &right_real_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &right_real_send, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &left_imag_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &left_imag_send, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &right_imag_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &right_imag_send, height * width * sizeof(double), hipHostMallocDefault));

    height = halo_y;	// The vertical halo in rows
    width = tile_width;	// The number of columns of the matrix
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &bottom_real_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &bottom_real_send, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &top_real_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &top_real_send, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &bottom_imag_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &bottom_imag_send, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &top_imag_receive, height * width * sizeof(double), hipHostMallocDefault));
    CUDA_SAFE_CALL(hipHostAlloc( (void **) &top_imag_send, height * width * sizeof(double), hipHostMallocDefault));

}


CC2Kernel::~CC2Kernel() {
    CUDA_SAFE_CALL(hipHostFree(left_real_receive));
    CUDA_SAFE_CALL(hipHostFree(left_real_send));
    CUDA_SAFE_CALL(hipHostFree(right_real_receive));
    CUDA_SAFE_CALL(hipHostFree(right_real_send));
    CUDA_SAFE_CALL(hipHostFree(bottom_real_receive));
    CUDA_SAFE_CALL(hipHostFree(bottom_real_send));
    CUDA_SAFE_CALL(hipHostFree(top_real_receive));
    CUDA_SAFE_CALL(hipHostFree(top_real_send));
    CUDA_SAFE_CALL(hipHostFree(left_imag_receive));
    CUDA_SAFE_CALL(hipHostFree(left_imag_send));
    CUDA_SAFE_CALL(hipHostFree(right_imag_receive));
    CUDA_SAFE_CALL(hipHostFree(right_imag_send));
    CUDA_SAFE_CALL(hipHostFree(bottom_imag_receive));
    CUDA_SAFE_CALL(hipHostFree(bottom_imag_send));
    CUDA_SAFE_CALL(hipHostFree(top_imag_receive));
    CUDA_SAFE_CALL(hipHostFree(top_imag_send));

    CUDA_SAFE_CALL(hipFree(pdev_real[0]));
    CUDA_SAFE_CALL(hipFree(pdev_real[1]));
    CUDA_SAFE_CALL(hipFree(pdev_imag[0]));
    CUDA_SAFE_CALL(hipFree(pdev_imag[1]));

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

void CC2Kernel::run_kernel_on_halo() {
    int inner = 0, horizontal = 0, vertical = 0;
    inner = 0;
    horizontal = 1;
    vertical = 0;
    numBlocks.x = (tile_width  + (BLOCK_X - 2 * halo_x) - 1) / (BLOCK_X - 2 * halo_x);
    numBlocks.y = 2;
    if(imag_time)
        imag_cc2kernel <<< numBlocks, threadsPerBlock, 0, stream1>>>(tile_width, tile_height, 0, 0, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real[sense], pdev_imag[sense], pdev_real[1 - sense], pdev_imag[1 - sense], inner, horizontal, vertical);
    else
        cc2kernel <<< numBlocks, threadsPerBlock, 0, stream1>>>(tile_width, tile_height, 0, 0, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real[sense], pdev_imag[sense], pdev_real[1 - sense], pdev_imag[1 - sense], inner, horizontal, vertical);

    inner = 0;
    horizontal = 0;
    vertical = 1;
    numBlocks.x = 2;
    numBlocks.y = (tile_height  + (BLOCK_Y - 2 * halo_y) - 1) / (BLOCK_Y - 2 * halo_y);
    if(imag_time)
        imag_cc2kernel <<< numBlocks, threadsPerBlock, 0, stream1>>>(tile_width, tile_height, 0, 0, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real[sense], pdev_imag[sense], pdev_real[1 - sense], pdev_imag[1 - sense], inner, horizontal, vertical);
    else
        cc2kernel <<< numBlocks, threadsPerBlock, 0, stream1>>>(tile_width, tile_height, 0, 0, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real[sense], pdev_imag[sense], pdev_real[1 - sense], pdev_imag[1 - sense], inner, horizontal, vertical);
}

void CC2Kernel::run_kernel() {
    int inner = 0, horizontal = 0, vertical = 0;
    inner = 1;
    horizontal = 0;
    vertical = 0;
    numBlocks.x = (tile_width  + (BLOCK_X - 2 * halo_x) - 1) / (BLOCK_X - 2 * halo_x) ;
    numBlocks.y = (tile_height + (BLOCK_Y - 2 * halo_y) - 1) / (BLOCK_Y - 2 * halo_y) - 2;

    if(imag_time)
        imag_cc2kernel <<< numBlocks, threadsPerBlock, 0, stream1>>>(tile_width, tile_height, 0, 0, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real[sense], pdev_imag[sense], pdev_real[1 - sense], pdev_imag[1 - sense], inner, horizontal, vertical);
    else
        cc2kernel <<< numBlocks, threadsPerBlock, 0, stream2>>>(tile_width, tile_height, 0, 0, halo_x, halo_y, a, b, dev_external_pot_real, dev_external_pot_imag, pdev_real[sense], pdev_imag[sense], pdev_real[1 - sense], pdev_imag[1 - sense], inner, horizontal, vertical);
    sense = 1 - sense;
    CUT_CHECK_ERROR("Kernel error in CC2Kernel::run_kernel");
}

void CC2Kernel::wait_for_completion(int iteration) {
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    //normalization
    if(imag_time && ((iteration % 20) == 0)) {

        CUDA_SAFE_CALL(hipMemcpy(p_real, pdev_real[sense], tile_width * tile_height * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(p_imag, pdev_imag[sense], tile_width * tile_height * sizeof(double), hipMemcpyDeviceToHost));

        int nProcs = 1;
#ifdef HAVE_MPI
        MPI_Comm_size(cartcomm, &nProcs);
#endif
        int height = tile_height - halo_y;
        int width = tile_width - halo_x;
        double sum = 0., sums[nProcs];
        for(int i = halo_y; i < height; i++) {
            for(int j = halo_x; j < width; j++) {
                sum += p_real[j + i * tile_width] * p_real[j + i * tile_width] + p_imag[j + i * tile_width] * p_imag[j + i * tile_width];
            }
        }
#ifdef HAVE_MPI
        MPI_Allgather(&sum, 1, MPI_DOUBLE, sums, 1, MPI_DOUBLE, cartcomm);
#else
        sums[0] = sum;
#endif
        double tot_sum = 0.;
        for(int i = 0; i < nProcs; i++)
            tot_sum += sums[i];
        double norm = sqrt(tot_sum);

        for(int i = 0; i < tile_height; i++) {
            for(int j = 0; j < tile_width; j++) {
                p_real[j + i * tile_width] /= norm;
                p_imag[j + i * tile_width] /= norm;
            }
        }
        CUDA_SAFE_CALL(hipMemcpy(pdev_real[sense], p_real, tile_width * tile_height * sizeof(double), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(pdev_imag[sense], p_imag, tile_width * tile_height * sizeof(double), hipMemcpyHostToDevice));
    }
}

void CC2Kernel::copy_results() {
    CUDA_SAFE_CALL(hipMemcpy(p_real, pdev_real[sense], tile_width * tile_height * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(p_imag, pdev_imag[sense], tile_width * tile_height * sizeof(double), hipMemcpyDeviceToHost));
}

void CC2Kernel::get_sample(size_t dest_stride, size_t x, size_t y, size_t width, size_t height, double * dest_real, double * dest_imag) const {
    assert(x < tile_width);
    assert(y < tile_height);
    assert(x + width <= tile_width);
    assert(y + height <= tile_height);
    CUDA_SAFE_CALL(hipMemcpy2D(dest_real, dest_stride * sizeof(double), &(pdev_real[sense][y * tile_width + x]), tile_width * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy2D(dest_imag, dest_stride * sizeof(double), &(pdev_imag[sense][y * tile_width + x]), tile_width * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost));
}

void CC2Kernel::start_halo_exchange() {

}

void CC2Kernel::finish_halo_exchange() {
#ifdef HAVE_MPI
    MPI_Request req[8];
    MPI_Status statuses[8];
#endif
    int offset = 0;

    // Halo copy: LEFT/RIGHT
    int height = inner_end_y - inner_start_y;	// The vertical halo in rows
    int width = halo_x;	// The number of columns of the matrix
    int stride = tile_width;	// The combined width of the matrix with the halo
    offset = (inner_start_y - start_y) * tile_width + inner_end_x - halo_x - start_x;
    CUDA_SAFE_CALL(hipMemcpy2DAsync(right_real_send, width * sizeof(double), &(pdev_real[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));
    CUDA_SAFE_CALL(hipMemcpy2DAsync(right_imag_send, width * sizeof(double), &(pdev_imag[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));
    offset = (inner_start_y - start_y) * tile_width + halo_x;
    CUDA_SAFE_CALL(hipMemcpy2DAsync(left_real_send, width * sizeof(double), &(pdev_real[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));
    CUDA_SAFE_CALL(hipMemcpy2DAsync(left_imag_send, width * sizeof(double), &(pdev_imag[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));

    // Halo copy: UP/DOWN
    height = halo_y;	// The vertical halo in rows
    width = tile_width;	// The number of columns of the matrix
    stride = tile_width;	// The combined width of the matrix with the halo

    offset = (inner_end_y - halo_y - start_y) * tile_width;
    CUDA_SAFE_CALL(hipMemcpy2DAsync(bottom_real_send, width * sizeof(double), &(pdev_real[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));
    CUDA_SAFE_CALL(hipMemcpy2DAsync(bottom_imag_send, width * sizeof(double), &(pdev_imag[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));
    offset = halo_y * tile_width;
    CUDA_SAFE_CALL(hipMemcpy2DAsync(top_real_send, width * sizeof(double), &(pdev_real[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));
    CUDA_SAFE_CALL(hipMemcpy2DAsync(top_imag_send, width * sizeof(double), &(pdev_imag[sense][offset]), stride * sizeof(double), width * sizeof(double), height, hipMemcpyDeviceToHost, stream1));

    hipStreamSynchronize(stream1);


    // Halo exchange: LEFT/RIGHT
    height = inner_end_y - inner_start_y;	// The vertical halo in rows
    width = halo_x;	// The number of columns of the matrix
    stride = tile_width;	// The combined width of the matrix with the halo

#ifdef HAVE_MPI
    MPI_Irecv(left_real_receive, height * width, MPI_DOUBLE, neighbors[LEFT], 1, cartcomm, req);
    MPI_Irecv(left_imag_receive, height * width, MPI_DOUBLE, neighbors[LEFT], 2, cartcomm, req + 1);
    MPI_Irecv(right_real_receive, height * width, MPI_DOUBLE, neighbors[RIGHT], 3, cartcomm, req + 2);
    MPI_Irecv(right_imag_receive, height * width, MPI_DOUBLE, neighbors[RIGHT], 4, cartcomm, req + 3);

    offset = (inner_start_y - start_y) * tile_width + inner_end_x - halo_x - start_x;
    MPI_Isend(right_real_send, height * width, MPI_DOUBLE, neighbors[RIGHT], 1, cartcomm, req + 4);
    MPI_Isend(right_imag_send, height * width, MPI_DOUBLE, neighbors[RIGHT], 2, cartcomm, req + 5);

    offset = (inner_start_y - start_y) * tile_width + halo_x;
    MPI_Isend(left_real_send, height * width, MPI_DOUBLE, neighbors[LEFT], 3, cartcomm, req + 6);
    MPI_Isend(left_imag_send, height * width, MPI_DOUBLE, neighbors[LEFT], 4, cartcomm, req + 7);

    MPI_Waitall(8, req, statuses);
#else
    if(periods[1] != 0) {
        memcpy2D(left_real_receive, height * width * sizeof(double), right_real_send, height * width * sizeof(double), height * width * sizeof(double), 1);
        memcpy2D(left_imag_receive, height * width * sizeof(double), right_imag_send, height * width * sizeof(double), height * width * sizeof(double), 1);
        memcpy2D(right_real_receive, height * width * sizeof(double) , left_real_send, height * width * sizeof(double), height * width * sizeof(double), 1);
        memcpy2D(right_imag_receive, height * width * sizeof(double) , left_imag_send, height * width * sizeof(double), height * width * sizeof(double), 1);
    }
#endif

    // Halo exchange: UP/DOWN
    height = halo_y;	// The vertical halo in rows
    width = tile_width;	// The number of columns of the matrix
    stride = tile_width;	// The combined width of the matrix with the halo

#ifdef HAVE_MPI
    MPI_Irecv(top_real_receive, height * width, MPI_DOUBLE, neighbors[UP], 1, cartcomm, req);
    MPI_Irecv(top_imag_receive, height * width, MPI_DOUBLE, neighbors[UP], 2, cartcomm, req + 1);
    MPI_Irecv(bottom_real_receive, height * width, MPI_DOUBLE, neighbors[DOWN], 3, cartcomm, req + 2);
    MPI_Irecv(bottom_imag_receive, height * width, MPI_DOUBLE, neighbors[DOWN], 4, cartcomm, req + 3);

    offset = (inner_end_y - halo_y - start_y) * tile_width;
    MPI_Isend(bottom_real_send, height * width, MPI_DOUBLE, neighbors[DOWN], 1, cartcomm, req + 4);
    MPI_Isend(bottom_imag_send, height * width, MPI_DOUBLE, neighbors[DOWN], 2, cartcomm, req + 5);

    offset = halo_y * tile_width;
    MPI_Isend(top_real_send, height * width, MPI_DOUBLE, neighbors[UP], 3, cartcomm, req + 6);
    MPI_Isend(top_imag_send, height * width, MPI_DOUBLE, neighbors[UP], 4, cartcomm, req + 7);

    MPI_Waitall(8, req, statuses);
#else
    if(periods[0] != 0) {
        memcpy2D(top_real_receive, height * width * sizeof(double), bottom_real_send, height * width  * sizeof(double), height * width * sizeof(double), 1);
        memcpy2D(top_imag_receive, height * width * sizeof(double), bottom_imag_send, height * width * sizeof(double), height * width * sizeof(double), 1);
        memcpy2D(bottom_real_receive, height * width * sizeof(double) , top_real_send, height * width * sizeof(double) , height * width * sizeof(double), 1);
        memcpy2D(bottom_imag_receive, height * width  * sizeof(double), top_imag_send, height * width * sizeof(double) , height * width * sizeof(double), 1);
    }
#endif
    // Copy back the halos to the GPU memory

    height = inner_end_y - inner_start_y;	// The vertical halo in rows
    width = halo_x;	// The number of columns of the matrix
    stride = tile_width;	// The combined width of the matrix with the halo

    offset = (inner_start_y - start_y) * tile_width;
    if (neighbors[LEFT] >= 0) {
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_real[sense][offset]), stride * sizeof(double), left_real_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_imag[sense][offset]), stride * sizeof(double), left_imag_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
    }
    offset = (inner_start_y - start_y) * tile_width + inner_end_x - start_x;
    if (neighbors[RIGHT] >= 0) {
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_real[sense][offset]), stride * sizeof(double), right_real_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_imag[sense][offset]), stride * sizeof(double), right_imag_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
    }

    height = halo_y;	// The vertical halo in rows
    width = tile_width;	// The number of columns of the matrix
    stride = tile_width;	// The combined width of the matrix with the halo

    offset = 0;
    if (neighbors[UP] >= 0) {
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_real[sense][offset]), stride * sizeof(double), top_real_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_imag[sense][offset]), stride * sizeof(double), top_imag_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
    }

    offset = (inner_end_y - start_y) * tile_width;
    if (neighbors[DOWN] >= 0) {
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_real[sense][offset]), stride * sizeof(double), bottom_real_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
        CUDA_SAFE_CALL(hipMemcpy2DAsync(&(pdev_imag[sense][offset]), stride * sizeof(double), bottom_imag_receive, width * sizeof(double), width * sizeof(double), height, hipMemcpyHostToDevice, stream1));
    }
}

